// Various invocation methods for libsort
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <algorithm>

#include "sort.h"
#include "utils.h"
/* #include "pyplover.h" */

// Sort provided input (h_in) in-place using the GPU
// Returns success status
extern "C" bool providedGpu(unsigned int* h_in, size_t len)
{
    SortState state (h_in, len);
    state.Step(0, 32);
    state.GetResult(h_in);
    return true;
}

// Sort provided input (in) using the CPU
// returns success status
extern "C" bool providedCpu(unsigned int* in, size_t len) {
    std::sort(in, in + len);
    return true;
}

// This function can be called by PyPlover as a KaaS function.
/* extern "C" void kaasInvoke(state_t *s, int grid, int block) { */
/*     radix_sort(s->out.dat, s->in.dat, s->in.len / sizeof(unsigned int)); */
/* } */
