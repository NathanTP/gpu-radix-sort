// Various invocation methods for libsort
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <algorithm>

#include "sort.h"
#include "utils.h"
/* #include "pyplover.h" */

// Sort provided input (h_in) in-place using the GPU
// Returns success status
extern "C" bool providedGpu(unsigned int* h_in, size_t len)
{
    // radix_sort is not in-place on the device so we have a temporary output array
    unsigned int* d_in;
    unsigned int* d_out;

    checkCudaErrors(hipMalloc(&d_in, sizeof(unsigned int) * len));
    checkCudaErrors(hipMalloc(&d_out, sizeof(unsigned int) * len));
    checkCudaErrors(hipMemcpy(d_in, h_in, sizeof(unsigned int) * len, hipMemcpyHostToDevice));
    radix_sort(d_out, d_in, len);
    checkCudaErrors(hipMemcpy(h_in, d_out, sizeof(unsigned int) * len, hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(d_out));
    checkCudaErrors(hipFree(d_in));

    return true;
}

// Sort provided input (in) using the CPU
// returns success status
extern "C" bool providedCpu(unsigned int* in, size_t len) {
    std::sort(in, in + len);
    return true;
}

// This function can be called by PyPlover as a KaaS function.
/* extern "C" void kaasInvoke(state_t *s, int grid, int block) { */
/*     radix_sort(s->out.dat, s->in.dat, s->in.len / sizeof(unsigned int)); */
/* } */
