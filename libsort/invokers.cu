// Various invocation methods for libsort
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <atomic>

#include "sort.h"
#include "utils.h"

// Perform a partial sort of bits [offset, width). boundaries will contain the
// index of the first element of each unique group value (each unique value of
// width bits), it must be 2^width elements long.
extern "C" bool gpuPartial(uint32_t* h_in, uint32_t *boundaries, size_t h_in_len, uint32_t offset, uint32_t width) {
    //auto-releases the reservation (if any) on destruction
    auto ctx = std::make_unique<cudaReservation>();
    if(!ctx->reserveDevice()) {
      return false;
    }

    //The sort internally only supports 32bit sizes
    if(h_in_len > UINT32_MAX) {
      fprintf(stderr, "Input array length must be less than 2^32\n");
      return false;
    }
    SortState state (h_in, h_in_len);

    state.Step(offset, width);
    state.GetResult(h_in);
    state.GetBoundaries(boundaries, offset, width);

    return true;
}

// Sort provided input (h_in) in-place using the GPU
// Returns success status
extern "C" bool providedGpu(unsigned int* h_in, size_t h_in_len)
{
    //auto-releases the reservation (if any) on destruction
    auto ctx = std::make_unique<cudaReservation>();
    if(!ctx->reserveDevice()) {
      return false;
    }

    //The sort internally only supports 32bit sizes
    if(h_in_len > UINT32_MAX) {
      fprintf(stderr, "Input array length must be less than 2^32\n");
      return false;
    }
    SortState state(h_in, h_in_len);

    state.Step(0, 32);
    state.GetResult(h_in);

    return true;
}

// Sort provided input (in) using the CPU
// returns success status
extern "C" bool providedCpu(unsigned int* in, size_t len) {
    std::sort(in, in + len);
    return true;
}
