// Various invocation methods for libsort
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <algorithm>

#include "sort.h"
#include "utils.h"
/* #include "pyplover.h" */

#define STEP_WIDTH 4
#define STEP_SIZE (1 << STEP_WIDTH)


// Perform a partial sort of bits [offset, width). boundaries will contain the
// index of the first element of each unique group value (each unique value of
// width bits), it must be 2^width elements long.
extern "C" bool gpuPartial(uint32_t* h_in, uint32_t *boundaries, size_t h_in_len, uint32_t offset, uint32_t width) {
    //The sort internally only supports 32bit sizes
    if(h_in_len > UINT32_MAX) {
      fprintf(stderr, "Input array length must be less than 2^32\n");
      return false;
    }
    SortState state (h_in, h_in_len);

    state.Step(offset, width);
    state.GetResult(h_in);
    state.GetBoundaries(boundaries, offset, width);
    return true;
}

// Sort provided input (h_in) in-place using the GPU
// Returns success status
extern "C" bool providedGpu(unsigned int* h_in, size_t h_in_len)
{
    //The sort internally only supports 32bit sizes
    if(h_in_len > UINT32_MAX) {
      fprintf(stderr, "Input array length must be less than 2^32\n");
      return false;
    }
    SortState state(h_in, h_in_len);

    state.Step(0, 32);
    state.GetResult(h_in);

    return true;
}

// Sort provided input (in) using the CPU
// returns success status
extern "C" bool providedCpu(unsigned int* in, size_t len) {
    std::sort(in, in + len);
    return true;
}

// This function can be called by PyPlover as a KaaS function.
/* extern "C" void kaasInvoke(state_t *s, int grid, int block) { */
/*     radix_sort(s->out.dat, s->in.dat, s->in.len / sizeof(unsigned int)); */
/* } */
